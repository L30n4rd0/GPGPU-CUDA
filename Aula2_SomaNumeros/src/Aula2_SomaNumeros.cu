/*
 ============================================================================
 Name        : ExemploSlide2.cu
 Author      : Leonardo
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void add(int a, int b, int *c) {
	c[0] = a + b;
}

int main(void) {
	int c;
	int *dev_c;

	hipMalloc((void**) &dev_c, sizeof(int));

	add<<<1, 1>>>(2, 7, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("2 + 7 = %d\n", c);

	hipFree(dev_c);

	return 0;
}
